#include "hip/hip_runtime.h"
//#include "gpuKernel.cc"
#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include "gpuKernelStupido.h"

#include <cmath>

// CUDA include files
#include <hip/hip_runtime.h>
//#include "TrackingTools/TransientTrack/interface/TransientTrack.h"



#define N   1024
#define RADIUS 3
#define BLOCK_SIZE 16

namespace gpuKernelStupido {
    
    __global__ void myKernel(int *in, int *out) {
            __shared__ int temp[BLOCK_SIZE + 2 * RADIUS]; 
            int gindex = threadIdx.x + blockIdx.x * blockDim.x;
            int lindex = threadIdx.x + RADIUS;

            // Read input elements into shared memory
            temp[lindex] = in[gindex];
            if (threadIdx.x < RADIUS) {
                temp[lindex - RADIUS] = in[gindex-RADIUS];
                temp[lindex + BLOCK_SIZE] = in[gindex+BLOCK_SIZE];
            }

            __syncthreads();


            int result = 0;

            for (int offset = -RADIUS; offset <= RADIUS; offset ++)
                result += temp[lindex+offset];

            out[gindex] = result;

    }

    void fill_ints(int * x, int n){
            std::fill_n(x, n, 1);
    }

    void makeAsync(track_SoA tks_SoA) {
        std::cout << "Ciao" << std::endl;         

        int *in, * out;
        int *d_in, *d_out;
        int size = (N+2*RADIUS) * sizeof(int);
        
        in = (int * ) malloc(size); fill_ints(in, N+ 2*RADIUS);
        out = (int *) malloc(size); fill_ints(out, N+ 2*RADIUS);
        
        hipMalloc((void **) &d_in, size);
        hipMalloc((void **) &d_out, size);
        
        hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);
        hipMemcpy(d_out, out, size, hipMemcpyHostToDevice);

        myKernel<<<N/BLOCK_SIZE, BLOCK_SIZE>>>(d_in + RADIUS, d_out + RADIUS);

        hipMemcpy(out, d_out,size, hipMemcpyDeviceToHost);
        
        free(in); free(out);
        hipFree(d_in); hipFree(d_out);
        
    }
}
